#include "hip/hip_runtime.h"
/*
 *  Test for checking the algorithms that calculate the interval evaluation of an RNS number
 */

#include <stdio.h>
#include <iostream>
#include "../src/rnseval.cuh"
#include "tsthelper.cuh"
#include "logger.cuh"

/*
 *  Printing the error of the computed interval evaluation with respect
 *  to the exact relative value of an RNS number
 */
void printError(interval_ptr eval, er_float_ptr exact) {
    std::cout << "\neval_low  = ";
    er_print(&eval->low);
    std::cout << "\neval_upp  = ";
    er_print(&eval->upp);

    er_adjust(exact);
    if((er_cmp(&eval->low, exact) == 1) || (er_cmp(exact, &eval->upp) == 1)){
        std::cout << "\nerror = 100%. The RNS Interval Evaluation is wrong!\n";
    }
    else{
        er_float_ptr error = new er_float_t[1];
        er_sub(error, &eval->upp, &eval->low);
        er_div(error, error, exact);
        double derror;
        er_get_d(&derror, error);
        std::cout << "\nrel.error = " << (derror) << std::endl;
        delete error;
    }
}

void resetResult(interval_ptr eval){
    er_set_d(&eval->low, 0.0);
    er_set_d(&eval->upp, 0.0);
}

__global__ void resetResultCuda(interval_ptr eval) {
    cuda::er_set_d(&eval->low, 0.0);
    cuda::er_set_d(&eval->upp, 0.0);
}

/*
 * CUDA tests
 */
__global__ void testCudaEvalCompute(interval_ptr d_eval, int * d_number) {
    cuda::rns_eval_compute(&d_eval->low, &d_eval->upp, d_number);
}

__global__ void testCudaEvalFastCompute(interval_ptr d_eval, int * d_number) {
    cuda::rns_eval_compute_fast(&d_eval->low, &d_eval->upp, d_number);
}

__global__ void testCudaEvalComputeParallel(interval_ptr d_eval, int * d_number) {
    cuda::rns_eval_compute_parallel(&d_eval->low, &d_eval->upp, d_number);
}

int main() {
    rns_const_init();
    Logger::beginTestDescription(Logger::TEST_VERIFY_RNSEVAL);
    rns_const_print(true);
    Logger::printDash();
    rns_eval_const_print();
    Logger::endSection(true);
    Logger::printSpace();

    bool asc = true; //start with x = 0
    char c;

    int * number = new int[RNS_MODULI_SIZE];;
    int * d_number;
    interval_ptr eval = new interval_t; // host result
    interval_ptr d_eval; // device result
    er_float_ptr exact = new er_float_t[1];
    mpz_t binary;

    hipMalloc(&d_number, RNS_MODULI_SIZE * sizeof(int));
    hipMalloc(&d_eval, sizeof(interval_t));
    mpz_init(binary);

    std::cout << "Enter 'y' for start (or any key for exit): ";
    std::cin >> c;
    if (c == 'y') {
        for (int j = 0; j < RNS_MODULI_SIZE; j++) {
            if (asc)
                number[j] = 0;
            else
                number[j] = RNS_MODULI[j] - 1;
        }
        hipMemcpy(d_number, number, RNS_MODULI_SIZE * sizeof(int), hipMemcpyHostToDevice);

        while (c == 'y') {
            for (int j = 0; j < 10; j++) {
                printf("\n");
                Logger::printDash();

                rns_to_binary(binary, number);
                printf("\nnumber = %s", mpz_get_str(NULL, 10, binary));
                rns_fractional(exact, number);
                std::cout << "\nrelative = ";
                er_print(exact);
                Logger::printSpace();
                //-------------------------------------------
                printf("\n[CPU] rns_eval_compute: ");
                resetResult(eval);
                rns_eval_compute(&eval->low, &eval->upp, number);
                printError(eval, exact);
                //-------------------------------------------
                printf("\n[CPU] rns_eval_compute_fast: ");
                resetResult(eval);
                rns_eval_compute_fast(&eval->low, &eval->upp, number);
                printError(eval, exact);
                //-------------------------------------------
                printf("\n[CUDA] rns_eval_compute: ");
                resetResult(eval);
                resetResultCuda<<< 1, 1 >>>(d_eval);
                testCudaEvalCompute<<< 1, 1 >>>(d_eval, d_number);
                hipMemcpy(eval, d_eval, sizeof(interval_t), hipMemcpyDeviceToHost);
                printError(eval, exact);
                //-------------------------------------------
                printf("\n[CUDA] rns_eval_compute_fast: ");
                resetResult(eval);
                resetResultCuda<<< 1, 1 >>>(d_eval);
                testCudaEvalFastCompute<<< 1, 1 >>>(d_eval, d_number);
                hipMemcpy(eval, d_eval, sizeof(interval_t), hipMemcpyDeviceToHost);
                printError(eval, exact);
                //-------------------------------------------
                printf("\n[CUDA] rns_eval_compute_parallel: ");
                resetResult(eval);
                resetResultCuda<<< 1, 1 >>>(d_eval);
                testCudaEvalComputeParallel<<< 1, RNS_MODULI_SIZE >>>(d_eval, d_number);
                hipMemcpy(eval, d_eval, sizeof(interval_t), hipMemcpyDeviceToHost);
                printError(eval, exact);
                //-------------------------------------------

                for (int i = 0; i < RNS_MODULI_SIZE; i++) {
                    if (asc)
                        number[i] = (number[i] + 1) % RNS_MODULI[i];
                    else {
                        number[i] -= 1;
                        if (number[i] < 0)
                            number[i] += RNS_MODULI[i];
                    }
                }
                hipMemcpy(d_number, number, RNS_MODULI_SIZE * sizeof(int), hipMemcpyHostToDevice);
            }
            Logger::printDash();
            std::cout << "\n\nEnter 'y' for continue (or any key for exit): ";
            std::cin >> c;
        }
    }

    delete [] number;
    delete eval;
    hipFree(d_number);
    hipFree(d_eval);
    delete [] exact;
    mpz_clear(binary);

    //End logging
    Logger::endTestDescription();
    return 1;
}